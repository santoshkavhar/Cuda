#include<stdio.h>
#include<hip/hip_runtime.h>
//N from 0 to 7
//#define N 
const int N = 128;
__global__ void bubble(float* a) {
	// a[0] = 0; a[1] = 33; a[2] = 66... a[];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid < N)
		a[tid] = N- threadIdx.x;
	else
		a[tid] = 0;

	__syncthreads();

	// no. of threads in a block
	//int i= blockDim.x/ 2;
	int temp;
	for (int i = 0; i < N; i++) {
		// if odd iteration and odd thread
		// if iteration is even and even thread
		if ((i % 2 && tid % 2) || ((i + 1) % 2 && (tid + 1) % 2)) {
			//for 0th element, be careful
			if (tid && ( a[tid - 1] > a[tid]) ) {
				temp = a[tid];
				a[tid] = a[tid - 1];
				a[tid - 1] = temp;
			}
		}
		__syncthreads();
	}
}

int main() {
	// copy dev_a from device to host
	// c wil hold the mean
	// dev_a is created on the device and holds the numbers
	float* a;
	float* dev_a;

	//allocate memory on CPU side
	a = (float*)malloc(N * sizeof(float));

	//allocate memory on GPU side
	hipMalloc((void**)&dev_a, N * sizeof(float));

	bubble<<<(N + 127)/128, 128 >>> (dev_a);
	// copy mean to c 
	hipMemcpy(a, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);


	for (int i = 0; i < N; i++)
		printf("\n%f", a[i]);
	free(a);
	hipFree(dev_a);
}

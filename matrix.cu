#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<graphics.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include <hiprand.h>
#define max 70
const int noofthreads = 48;
const int noofblocks= 64;
__global__ void matrix(int *old, int *x,int *y){
	__shared__ count= 100;
	__shared__ int new[noofthreads];
	
	int tid= threadIdx.x;
	
	if(tid)
		new[threadIdx.x] = old[threadIdx.x + 1][blockIdx.x] % 5;
	
	else{
		
		
		new[threadIdx.x] =  % 5;
		// assume have already set up hiprand and generated state for each thread...
		// assume ranges vary by thread index
		
	}	
	__syncthreads();
	
	x[threadIdx.x][blockIdx.x] = threadIdx.x * 10;
	y[threadIdx.x][blockIdx.x] = blockIdx.x * 10;
	
	old[threadIdx.x][blockIdx.x] = new[threadIdx.x];
	
}

int main()
{
    int gd=DETECT, gm;
	//int *xii, *yii;
    int xi[noofthreads][noofblocks], yi[noofthreads][noofblocks];
    int r[noofthreads][noofblocks] ; // r for position of 1 or 0.
    //int cnt=0, x, y;    //cnt will help for randomly filling the first cycle of whole screen
	
    initgraph(&gd, &gm, NULL);
    
    int *dev_a, *dev_x, dev_y;
    
    hipMalloc((void**)&dev_a,  noofthreads * noofblocks * sizeof(int));
    hipMalloc((void**)&dev_x,  noofthreads * noofblocks * sizeof(int));
    hipMalloc((void**)&dev_y,  noofthreads * noofblocks * sizeof(int));
    
    hipMemcpy(dev_a, r, noofthreads * noofblocks * sizeof(int), hipMemcpyHostToDevice );
	
	matrix<<<noofblocks,noofthreads>>>(dev_a, xi, yi);
	
	hipMemcpy(r, dev_a, noofthreads * noofblocks * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy(xi, dev_x, noofthreads * noofblocks * sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy(yi, dev_y, noofthreads * noofblocks * sizeof(int), hipMemcpyDeviceToHost );
    
    
    
do{
	
	hipMemcpy(dev_a, r, noofthreads * noofblocks * sizeof(int), hipMemcpyHostToDevice );
	
	matrix<<<noofblocks,noofthreads>>>(dev_a, xi, yi);
	
	hipMemcpy(r, dev_a, noofthreads * noofblocks * sizeof(int), hipMemcpyDeviceToHost );
	
     for(int i=0; i<noofblocks; i++){
          for(int j=0; j<noofthreads; j+=2){    //extra space needed to look neat, so j is double incremented
                        
               // x= i*10 ;       //as the window is divided into 64*48, we multiply by 10 for position
               // y= j*10 ;
                                
                        /// This is for making that position blank and ready for next entry
                setcolor(0);    // make it disapper
                                            
                if(r[i][j]==0){ 
                       line(xi[i][j] + 4, yi[i][j]  + 6, xi[i][j]  + 5, yi[i][j]  + 5);//1's head
                       line (xi[i][j]  + 5, yi[i][j]  + 5, xi[i][j]  + 5, yi[i][j]  + 15);// 1's spine
                       line(xi[i][j]  + 4, yi[i][j]  + 15, xi[i][j]  + 6, yi[i][j]  + 15);// 1's base
                }
                else if(r[i][j]==1)
                       ellipse(xi[i][j]  + 5, yi[i][j]  + 10, 0, 360, 2 , 5); //draw 0
                 
 //delay(10);
                setcolor(2);
                               
                if(r[i][j]==0){
                      line(xi[i][j]  + 4, yi[i][j]  + 6, xi[i][j]  + 5, yi[i][j]  + 5);
                      line (xi[i][j]  + 5, yi[i][j]  + 5, xi[i][j]  + 5, yi[i][j]  + 15);
                      line(xi[i][j]  + 4, yi[i][j]  + 15, xi[i][j]  + 6, yi[i][j]  + 15);
                }
                else if(r[i][j]==1)
                      ellipse(xi[i][j]  + 5, yi[i][j]  + 10, 0, 360, 2 , 5);
                    }
                }
}while(1);		//for infinite loop
           //delay(9999);
    closegraph();
}

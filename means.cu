#include<stdio.h>
#include<hip/hip_runtime.h>
#define sum_numbers(x) (x*(x+1)/2)
//N from 0 to 7
#define N 13000	
const int NoOfThreads = 128;
const int NoOfBlocks = (N+127)/128;

__global__ void means(float *a){
    // a[0] = 0; a[1] = 1;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N)
        a[id] = id;
	else
		a[id] = 0;
    __syncthreads();
    // no. of threads in a block
    
    int i= blockDim.x/2;
    while(i != 0){
        if(threadIdx.x < i)
            a[id] += a[id + i];
        __syncthreads();
        i /= 2;
    }
    __syncthreads();
	// 0th thread of each block except 0th block add their sum to a[0] atomically
    //threadIdx.x =0 && id != 0
    if(!threadIdx.x && id)
    	atomicAdd(&a[0], a[id] );
   //cudaDeviceSynchronize();
   
   	//only 0th thread of 0th block computes mean
   	// it doesn't work!!!
   	// send directly the total sum
}


int main(){
    // copy dev_a from device to host
    // c wil hold the mean
    // dev_a is created on the device and holds the numbers
    float c; /* *a,*/
    float *dev_a;
	//Lock lock;
    //allocate memory on CPU side
    //a = (float*)malloc(N*sizeof(float));

    //allocate memory on GPU side
    hipMalloc( (void**)&dev_a, N*sizeof(float));
	
    means<<<NoOfBlocks,NoOfThreads>>>(dev_a);
    // copy mean to c 
    hipMemcpy(&c, dev_a, sizeof(float), hipMemcpyDeviceToHost);
    // computing mean on CPU than GPU because of error caused
    // GPU doesn't have block synchronisation so other blocks sum isn't written to 0th block
    c= c/N;
    printf("Does GPU value %.6g = %.6g\n", c, (sum_numbers( (float)(N - 1) )/N) );
    
    hipFree(dev_a);
    
    return 0;
}
